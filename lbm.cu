#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include "lbm.h"

#define F(x) f[size * x + index]
#define NEW_F(x) new_f[size * x + index]

__global__ void calcBoundary(int *boundary, const bool *obstacles, const int width, const int height)
{
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int index = col + row * width;

    if (col >= width || row >= height || obstacles[index])
    {
        return;
    }

    // determine boundary based on neighboring obstacles

    // define an array of directions: horizontal, vertical, main diagonal, secondary diagonal
    const int dirs[4][2] = {{1, 0}, {0, 1}, {1, 1}, {-1, 1}};
    const int size = width * height;

    // loop through each direction
    for (int d = 0; d < 4; d++)
    {
        // get the offsets for the current direction
        const int dx = dirs[d][0];
        const int dy = dirs[d][1];

        // check the adjacent cells in the current direction
        if (col - dx >= 0 && row - dy >= 0 && obstacles[col - dx + (row - dy) * width])
        {
            boundary[size * d + index] = -1;
        }
        else if (col + dx < width && row + dy < height && obstacles[col + dx + (row + dy) * width])
        {
            boundary[size * d + index] = 1;
        }
        else
        {
            boundary[size * d + index] = 0;
        }
    }
}

__global__ void init(float *f, float *rho, float *ux, float *uy, const int width, const int height,
                     const bool *obstacles)
{
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int index = col + row * width;

    if (col >= width || row >= height)
    {
        return;
    }
    if (obstacles[index])
    {
        // set ux uy to quiet nan
        ux[index] = nanf("");
        uy[index] = nanf("");
        return;
    }

    const int size = width * height;
    const float weights[9] = {4.0 / 9.0,  1.0 / 9.0,  1.0 / 9.0,  1.0 / 9.0, 1.0 / 9.0,
                              1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0};

    for (int i = 0; i < 9; i++)
    {
        F(i) = weights[i];
    }
    rho[index] = 1;
    ux[index] = 0;
    uy[index] = 0;
}

__global__ void step1(const int width, const int height, const int it, const float u_in_now, const float om_p,
                      const float sum_param, const float sub_param, float *f, float *new_f, float *rho, float *ux,
                      float *uy, float *u_out, const int *boundary, const bool *obstacle)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int index = row * width + col;

    // return if out of bounds or obstacle
    if (row >= height || col >= width || obstacle[index])
        return;

    const int size = width * height;
    const int velocitiesX[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
    const int velocitiesY[9] = {0, 0, -1, 0, 1, -1, -1, 1, 1};
    const float weights[9] = {4.0 / 9.0,  1.0 / 9.0,  1.0 / 9.0,  1.0 / 9.0, 1.0 / 9.0,
                              1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0};
    const int opposite[9] = {0, 3, 4, 1, 2, 7, 8, 5, 6};

    // if i'm a any boundary set u to 0
    if (row == 0 || row == height - 1 || col == 0 || col == width - 1)
    {
        ux[index] = 0;
        uy[index] = 0;
    }

    // set parabolic profile inlet
    if (col == 0)
    {
        const float halfDim = static_cast<float>(height - 1) / 2.0;
        const float temp = static_cast<float>(row / halfDim) - 1.0;
        const float mul = 1.0 - temp * temp;
        ux[index] = u_in_now * mul;
    }

    // zou he

    // top wall
    if (row == 0 && col != 0 && col != width - 1)
    {
        rho[index] = (F(0) + F(1) + F(3) + 2.0 * (F(2) + F(5) + F(6))) / (1.0 + uy[index]);
        F(4) = F(2) - 2.0 / 3.0 * rho[index] * uy[index];
        F(7) = F(5) + 0.5 * (F(1) - F(3)) - 0.5 * rho[index] * ux[index] - 1.0 / 6.0 * rho[index] * uy[index];
        F(8) = F(6) - 0.5 * (F(1) - F(3)) + 0.5 * rho[index] * ux[index] - 1.0 / 6.0 * rho[index] * uy[index];
    }
    // right wall
    else if (col == width - 1 && row != 0 && row != height - 1)
    {
        rho[index] = 1;
        ux[index] = F(0) + F(2) + F(4) + 2.0 * (F(1) + F(5) + F(8)) - 1.0;
        F(3) = F(1) - 2.0 / 3.0 * ux[index];
        F(6) = F(8) - 0.5 * (F(2) - F(4)) - 1.0 / 6.0 * ux[index];
        F(7) = F(5) + 0.5 * (F(2) - F(4)) - 1.0 / 6.0 * ux[index];
    }
    // bottom wall
    else if (row == height - 1 && col != 0 && col != width - 1)
    {
        rho[index] = (F(0) + F(1) + F(3) + 2.0 * (F(4) + F(7) + F(8))) / (1.0 - uy[index]);
        F(2) = F(4) + 2.0 / 3.0 * rho[index] * uy[index];
        F(5) = F(7) - 0.5 * (F(1) - F(3)) + 0.5 * rho[index] * ux[index] + 1.0 / 6.0 * rho[index] * uy[index];
        F(6) = F(8) + 0.5 * (F(1) - F(3)) - 0.5 * rho[index] * ux[index] + 1.0 / 6.0 * rho[index] * uy[index];
    }
    // left wall
    else if (col == 0 && row != 0 && row != height - 1)
    {
        rho[index] = (F(0) + F(2) + F(4) + 2.0 * (F(3) + F(7) + F(6))) / (1.0 - ux[index]);
        F(1) = F(3) + 2.0 / 3.0 * rho[index] * ux[index];
        F(5) = F(7) - 0.5 * (F(2) - F(4)) + 1.0 / 6.0 * rho[index] * ux[index] + 0.5 * rho[index] * uy[index];
        F(8) = F(6) + 0.5 * (F(2) - F(4)) + 1.0 / 6.0 * rho[index] * ux[index] - 0.5 * rho[index] * uy[index];
    }
    // top right corner
    else if (row == 0 && col == width - 1)
    {
        rho[index] = rho[index - 1];
        F(3) = F(1) - 2.0 / 3.0 * rho[index] * ux[index];
        F(4) = F(2) - 2.0 / 3.0 * rho[index] * uy[index];
        F(7) = F(5) - 1.0 / 6.0 * rho[index] * ux[index] - 1.0 / 6.0 * rho[index] * uy[index];
        F(8) = 0;
        F(6) = 0;
        F(0) = rho[index] - F(1) - F(2) - F(3) - F(4) - F(5) - F(7);
    }
    // bottom right corner
    else if (row == height - 1 && col == width - 1)
    {
        rho[index] = rho[index - 1];
        F(3) = F(1) - 2.0 / 3.0 * rho[index] * ux[index];
        F(2) = F(4) + 2.0 / 3.0 * rho[index] * uy[index];
        F(6) = F(8) + 1.0 / 6.0 * rho[index] * uy[index] - 1.0 / 6.0 * rho[index] * ux[index];
        F(7) = 0;
        F(5) = 0;
        F(0) = rho[index] - F(1) - F(2) - F(3) - F(4) - F(6) - F(8);
    }
    // bottom left corner
    else if (row == height - 1 && col == 0)
    {
        rho[index] = rho[index + 1];
        F(1) = F(3) + 2.0 / 3.0 * rho[index] * ux[index];
        F(2) = F(4) + 2.0 / 3.0 * rho[index] * uy[index];
        F(5) = F(7) + 1.0 / 6.0 * rho[index] * ux[index] + 1.0 / 6.0 * rho[index] * uy[index];
        F(6) = 0;
        F(8) = 0;
        F(0) = rho[index] - F(1) - F(2) - F(3) - F(4) - F(5) - F(7);
    }
    // top left corner
    else if (row == 0 && col == 0)
    {
        rho[index] = rho[index + 1];
        F(1) = F(3) + 2.0 / 3.0 * rho[index] * ux[index];
        F(4) = F(2) - 2.0 / 3.0 * rho[index] * uy[index];
        F(8) = F(6) + 1.0 / 6.0 * rho[index] * ux[index] - 1.0 / 6.0 * rho[index] * uy[index];
        F(7) = 0;
        F(5) = 0;
        F(0) = rho[index] - F(1) - F(2) - F(3) - F(4) - F(6) - F(8);
    }

    // update macro

    rho[index] = 0;
    ux[index] = 0;
    uy[index] = 0;
    for (int i = 0; i < 9; i++)
    {
        rho[index] += F(i);
        ux[index] += F(i) * velocitiesX[i];
        uy[index] += F(i) * velocitiesY[i];
    }
    ux[index] /= rho[index];
    uy[index] /= rho[index];
    u_out[index] = sqrtf(ux[index] * ux[index] + uy[index] * uy[index]);

    // equilibrium
    float feq[9];
    const float temp1 = 1.5 * (ux[index] * ux[index] + uy[index] * uy[index]);
    for (int i = 0; i < 9; i++)
    {
        const float temp2 = 3.0 * (velocitiesX[i] * ux[index] + velocitiesY[i] * uy[index]);
        feq[i] = weights[i] * rho[index] * (1.0 + temp2 + 0.5 * temp2 * temp2 - temp1);
    }

    // collision for index 0
    NEW_F(0) = (1.0 - om_p) * F(0) + om_p * feq[0];

    // collision for other indices
    for (int i = 1; i < 9; i++)
    {
        NEW_F(i) =
            (1.0 - sum_param) * F(i) - sub_param * F(opposite[i]) + sum_param * feq[i] + sub_param * feq[opposite[i]];
    }

    // regular bounce back

    if (boundary[index] == 1)
    {
        F(3) = NEW_F(1);
    }
    else if (boundary[index] == -1)
    {
        F(1) = NEW_F(3);
    }
    if (boundary[size + index] == 1)
    {
        F(2) = NEW_F(4);
    }
    else if (boundary[size + index] == -1)
    {
        F(4) = NEW_F(2);
    }
    if (boundary[size * 2 + index] == 1)
    {
        F(6) = NEW_F(8);
    }
    else if (boundary[size * 2 + index] == -1)
    {
        F(8) = NEW_F(6);
    }
    if (boundary[size * 3 + index] == 1)
    {
        F(5) = NEW_F(7);
    }
    else if (boundary[size * 3 + index] == -1)
    {
        F(7) = NEW_F(5);
    }
}

__global__ void step2(const int width, const int height, float *f, const float *new_f, const bool *obstacle)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int index = row * width + col;

    // return if out of bounds or obstacle
    if (row >= height || col >= width || obstacle[index])
        return;

    const int size = width * height;
    const int velocitiesX[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
    const int velocitiesY[9] = {0, 0, -1, 0, 1, -1, -1, 1, 1};

    // stream for index 0
    F(0) = NEW_F(0);

    // stream for other indices
    for (int i = 1; i < 9; i++)
    {
        // obtain new indices
        const int new_row = row + velocitiesY[i];
        const int new_col = col + velocitiesX[i];
        const int new_index = new_row * width + new_col;
        // stream if new index is not out of bounds or obstacle
        if (new_row >= 0 && new_row < height && new_col >= 0 && new_col < width && !obstacle[new_index])
        {
            f[size * i + new_index] = NEW_F(i);
        }
    }
}


#ifdef AB_TESTING
	#define hipMalloc(ptr, size) hipMallocManaged(ptr, size)
#endif


int main(int argc, char *argv[]) {
	if (argc != 2) {
		std::cerr << "Invalid command line arguments" << std::endl;
		std::cerr << "Usage: lbm <output filename>"   << std::endl;

		return -1;
	}

	const char *output_filename = argv[1];

    // read input

    std::ifstream file_in("input.txt");
    std::ofstream file_out(output_filename, std::ios::binary);

    int width, height, max_it, it = 0;
    file_in >> width >> height;
    file_in.get();
    file_out << width << ' ' << height << '\n';

    float reynolds, u_in;
    file_in >> reynolds >> max_it >> u_in;
    file_in.get();

    // calculate simulation parameters

    const float nu = u_in * static_cast<float>(height) / reynolds * 2.0 / 3.0;
    const float tau = 3.0 * nu + 0.5;
    const float sigma = ceil(10.0 * height);
    const float double_square_sigma = 2.0 * sigma * sigma;
    const float lambda_trt = 1.0 / 4.0;
    const float tau_minus = lambda_trt / (tau - 0.5) + 0.5;
    const float omega_plus = 1.0 / tau;
    const float omega_minus = 1.0 / tau_minus;
    const float sub_param = 0.5 * (omega_plus - omega_minus);
    const float sum_param = 0.5 * (omega_plus + omega_minus);

    // allocate memory

    float *ux, *uy, *f, *new_f, *rho, *u_out, *u_out_host;
    int *host_boundary;
    int *boundary;
    bool *obstacles, *host_obstacles;

    hipHostMalloc(&host_obstacles, width * height * sizeof(bool));
    hipMalloc(&obstacles, width * height * sizeof(bool));
    hipMalloc(&ux, width * height * sizeof(float));
    hipMalloc(&uy, width * height * sizeof(float));
    hipMalloc(&u_out, width * height * sizeof(float));
    hipHostMalloc(&u_out_host, width * height * sizeof(float));
    hipMalloc(&f, width * height * 9 * sizeof(float));
    hipMalloc(&new_f, width * height * 9 * sizeof(float));
    hipMalloc(&rho, width * height * sizeof(float));

    hipMalloc(&boundary, width * height * 4 * sizeof(int));
    hipHostMalloc(&host_boundary, width * height * 4 * sizeof(int));

    // read obstacles

    hipMemset(host_obstacles, 0, width * height * sizeof(bool));
    while (file_in.peek() != EOF)
    {
        int x, y;
        file_in >> x >> y;
        host_obstacles[x + y * width] = true;
    }
    file_in.close();

    const dim3 threads_per_block(24, 24);
    const dim3 num_blocks(ceil(width / 24.0), ceil(height / 24.0));

    hipMemcpy(obstacles, host_obstacles, width * height * sizeof(bool), hipMemcpyHostToDevice);
    // hipHostFree(host_obstacles);

    // initialize values and create streams

    // calcBoundary<<<num_blocks, threads_per_block>>>(boundary, obstacles, width, height);
    lbm_calc_boundary(host_boundary, host_obstacles, width, height);
    hipMemcpy(boundary, host_boundary, width * height * 4 * sizeof(int), hipMemcpyHostToDevice);


    //init<<<num_blocks, threads_per_block>>>(f, rho, ux, uy, width, height, obstacles);
    hipDeviceSynchronize();
    lbm_init(f, rho, ux, uy, width, height, obstacles);
    hipDeviceSynchronize();



    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // main loop

    while (it <= max_it)
    {
        const float u_in_now = u_in * (1.0 - std::exp(-static_cast<double>(it * it) / double_square_sigma));

        step1<<<num_blocks, threads_per_block, 0, stream1>>>(width, height, it, u_in_now, omega_plus, sum_param,
                                                             sub_param, f, new_f, rho, ux, uy, u_out, boundary,
                                                             obstacles);
        hipStreamSynchronize(stream1);


        // step2<<<num_blocks, threads_per_block, 0, stream1>>>(width, height, f, new_f, obstacles);
	hipDeviceSynchronize();
	lbm_step2(width, height, f, new_f, obstacles);
	hipDeviceSynchronize();


        if (it % (max_it / 100) == 0)
        {
            // copy results to host using stream 2 and wait for it to finish
            hipMemcpyAsync(u_out_host, u_out, width * height * sizeof(float), hipMemcpyDeviceToHost, stream2);
            hipStreamSynchronize(stream2);

            // write results to file
		file_out << it << '\n';
		file_out.write(reinterpret_cast<char *>(u_out_host), sizeof(float) * width * height);

            // print to console
            std::cout << it << '\n';
        }

        it++;
    }
}
